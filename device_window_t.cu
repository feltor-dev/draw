#include "hip/hip_runtime.h"
//NOT WORKING WITH NEWER THRUST VERSION
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "device_window.cuh"
/**
 * @brief Functor returning a gaussian
 * \f[
   f(x,y) = Ae^{-(\frac{(x-x_0)^2}{2\sigma_x^2} + \frac{(y-y_0)^2}{2\sigma_y^2}}
   \f]
 */
struct Gaussian
{
    /**
     * @brief Functor returning a gaussian
     *
     * @param x0 x-center-coordinate
     * @param y0 y-center-coordinate
     * @param sigma_x x - variance
     * @param sigma_y y - variance
     * @param amp Amplitude
     */
    Gaussian( float x0, float y0, float sigma_x, float sigma_y, float amp)
        : x00(x0), y00(y0), sigma_x(sigma_x), sigma_y(sigma_y), amplitude(amp){}
    /**
     * @brief Return the value of the gaussian
     *
     * \f[
       f(x,y) = Ae^{-(\frac{(x-x_0)^2}{2\sigma_x^2} + \frac{(y-y_0)^2}{2\sigma_y^2}}
       \f]
     * @param x x - coordinate
     * @param y y - coordinate
     *
     * @return gaussian
     */
    float operator()(float x, float y)
    {
        return  amplitude*
                   exp( -((x-x00)*(x-x00)/2./sigma_x/sigma_x +
                          (y-y00)*(y-y00)/2./sigma_y/sigma_y) );
    }
  private:
    float  x00, y00, sigma_x, sigma_y, amplitude;

};

const unsigned Nx = 20, Ny = 10;
const float lx = 2., ly = 2.;
const float hx = lx/(float)Nx, hy = ly/(float)Ny;

int main()
{
    //Create Window and set window title
    GLFWwindow* w = draw::glfwInitAndCreateWindow( 800, 400, "Hello world!");
    draw::RenderDeviceData render( 1,1);
    // generate a vector on the grid to visualize
    Gaussian g( 1.2, 0.3, .1, .1, 1);
    thrust::host_vector<float> visual(Nx*Ny);
    for(unsigned i=0; i<Ny; i++)
        for( unsigned j=0; j<Nx; j++)
            visual[i*Nx+j] = g( (float)j*hx, (float)i*hy);
    thrust::device_vector<float> dvisual1 = visual;
    for(unsigned i=0; i<Ny; i++)
        for( unsigned j=0; j<Nx; j++)
            visual[i*Nx+j] = -g( (float)j*hx, (float)i*hy);
    thrust::device_vector<float> dvisual2 = visual;

    //create a colormap
    draw::ColorMapRedBlueExt colors( 1.);
    //set scale
    colors.scale() =  1.;

    while ( !glfwWindowShouldClose( w))
    {
        render.renderQuad( dvisual1, Nx, Ny, colors);
        //render.renderQuad( dvisual2, Nx, Ny, colors);
        glfwSwapBuffers(w);
        glfwWaitEvents();
    }

    return 0;
}
